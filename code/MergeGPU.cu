#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <hip/hip_runtime.h>


#define NTPB 64
#define NEPT 2
#define SIZE 128
#define MAX(x,y) ((x<=y)? y : x)
#define MIN(x,y) ((x<=y)? x : y)

// Function that catches the error 
void testCUDA(hipError_t error, const char *file, int line)  {

	if (error != hipSuccess) {
	   printf("There is an error in file %s at line %d\n", file, line);
       exit(EXIT_FAILURE);
	} 
}
// Has to be defined in the compilation in order to get the correct value of the macros
// __FILE__ and __LINE__
#define testCUDA(error) (testCUDA(error, __FILE__ , __LINE__))

void printGPUCaracteristics(){
	int count;
	hipDeviceProp_t prop;

	testCUDA(hipGetDeviceCount(&count));
	printf("\n\nThe number of devices available is %i GPUs \n", count);
	testCUDA(hipGetDeviceProperties(&prop, count-1));
	printf("Name: %s\n",  prop.name);
	printf("Global memory size in octet (bytes): %ld\n", prop.totalGlobalMem);
	printf("Shared memory size per block: %ld\n", prop.sharedMemPerBlock);
	printf("Number of registers per block: %i\n", prop.regsPerBlock);
	printf("Number of threads in a warp: %i\n", prop.warpSize);
	printf("Maximum number of threads that can be launched per block: %i\n", 
		   prop.maxThreadsPerBlock);
	printf("Maximum number of threads that can be launched: %i X %i X %i\n", 
		   prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("Maximum grid size: %i X %i X %i\n", prop.maxGridSize[0], 
		   prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("Total constant memory size: %ld\n", prop.totalConstMem);
	printf("Major compute capability: %i\n", prop.major);
	printf("Minor compute capability: %i\n", prop.minor);
	printf("Clock rate: %i\n", prop.clockRate);
	printf("Maximum 1D texture memory: %i\n", prop.maxTexture1D);
	printf("Could we overlap?: %i\n", prop.deviceOverlap);
	printf("Number of multiprocessors: %i\n", prop.multiProcessorCount);
	printf("Is there a limit for kernel execution?: %i\n", 
		   prop.kernelExecTimeoutEnabled);
	printf("Is my GPU a chipset?: %i\n", prop.integrated);
	printf("Can we map the host memory?: %i\n", prop.canMapHostMemory);
	printf("Can we launch concurrent kernels?: %i\n", prop.concurrentKernels);
	printf("Do we have ECC memory?: %i\n", prop.ECCEnabled);
}

/* Cette fonction fait :
* d'abord: chercher le point d'intersection de diagonal avec merge-path 
* et puis merge.
*
* paramètre:
* @ A, B: 2 tableaux triés
* @ C: tableau output qui contient A et B et qui est trié
* @ size_A, size_B : la taille du tableau A et B
* @ thread_id : indentifiant de thread dans la grille
* @ tid : indentifiant de thread dans son block
* @ numThreads : le nombre de threads dans la grille
*/
__device__ void DiagonalIntersection_Merge(int *A, int size_A ,int * B, int size_B, int *C, int thread_id, int tid, int numThreads)
{
	int diag, diaglength,  a_top, b_top, a_bottom, b_bottom;
	int offset, a_end=0, b_end=0, a_start, b_start;
	int numEls = (size_A + size_B)/numThreads;
	int c_start;
	__shared__ int diagA[NTPB];
	__shared__ int diagB[NTPB];

	/*chaque thread détermine sa propre matrice dont diagonal
	* contient le point d'intersection avec merge-path
	*/	
	diag = (thread_id) * numEls;
	a_top = (diag > size_A) ? size_A : diag;
	b_top = (diag > size_A) ? diag - size_A : 0;
	a_bottom = b_top;
	b_bottom = diaglength = (diag > size_A) ? 2*size_A - diag : diag;	
	/*initialiser tableaux diagA, diagB*/
	if (thread_id==0)
	{
		diaglength = 0.5;	
		a_start = diagA[tid] = a_bottom;
		b_start = diagB[tid] = b_top; 
	}
	else
	{
  		diagA[tid] = a_bottom;
		diagB[tid] = b_top + diaglength - 1; 
	}

	/*recherche dichotomique*/		
	while(diaglength > 0.5)
	{
		offset = (a_top - a_bottom)/2;
		a_start    = a_top - offset ;
		b_start    = b_top + offset ;

		/*des cas spécials où l'intersection est sur le bord de matrice A*B*/
		if (a_start == a_top && b_start == 0 | a_start == a_top && b_start == b_bottom | b_start == size_A)
		{
			diagB[tid] = b_start;
			diagA[tid] = a_start;
			break;
		}

		/*des cas réguliers où l'intersection est dans la matrice*/	
		if(A[a_start] > B[b_start-1])
		{
			if(A[a_start-1] <= B[b_start])
			{
				/*le point au milieu est celui d'intersection*/
				diagA[tid] = a_start;
				diagB[tid] = b_start;
				break;
			}
			else
			{
				/*on se deplace sur la partie plus petit*/
				/*redéterminer la nouvelle matrice*/
				a_top = a_start - 1;
				b_top = b_start + 1;
			}
		}
		else
		{
			/*on se deplace sur la partie plus grand*/
			/*redeterminer la nouvelle matrice*/
			a_bottom = a_start;
		}
		diaglength /= 2;
	}

	/*dans la suite, chaque va lire un élément de diagA, un élément de diagB.
	* Pour cela, des écritures dans ces tableaux doivent terminer avant la commence de lecturee. 
	*Faut synchoniser des threads.
	*/
	__syncthreads();

	/*chaque thread lit un élément de diagA et un de diagB 
	* pour determiner a_end, b_end
	*/
	if (tid < NTPB -1)
	{
		a_end = diagA[tid+1];
		b_end = diagB[tid+1];
	}
	else
	{
		/*des threads dont tid = NTPB-1 ne peuvent pas communiquer avec sa voisine à droite
		* car elles sont dans différents block. Pour telle thread, a_end, b_end sont determiné différemment
		*/
		if (thread_id < numThreads - 1)
		{
			a_end = a_start + numEls;
			b_end = b_start + numEls;
		}
		else
		{
			a_end = size_A;
			b_end = size_A;
		}
	}

	__syncthreads();

	/*Partie MERGE*/
	
	c_start = thread_id * numEls;
	
	int c_end = c_start + numEls;
    	while (a_start < a_end && b_start < b_end && c_start < c_end) {
        	if (A[a_start] <= B[b_start]) {
			C[c_start] = A[a_start];
			c_start++;
			a_start++;
        	} else {
			C[c_start] = B[b_start];
			c_start++;
			b_start++;
        	}
    	}

    	while(a_start < a_end  && c_start < c_end) {
		C[c_start] = A[a_start];
		c_start++;
		a_start++;
    	}

	while(b_start < b_end  && c_start < c_end) {
		C[c_start] = B[b_start];
		c_start++;
		b_start++;
    	}
	
}

/*la fonction kernel de chaque thread
*
*paramètre:
*@ A,B : tableaux d'entrée
*@ size_A, size_B : taille de A et B
*@ numThreads : nombre de threads dans la grille
*/
__global__ void kernel(int *A, int *B, int *S,int size_A, int size_B, int numThreads){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	DiagonalIntersection_Merge(A, size_A , B, size_B, S, idx, threadIdx.x, numThreads);
}

void printArray(int A[], int size){
    	int i;
    	for (i=0; i < size; i++){
        	printf("%d ", A[i]);
	}
    	printf("\n\n");
}


void wrapper(int *A, int *B, int *S, int size_A, int size_B){

	int *A_GPU , *B_GPU,*S_GPU;
	int size_S = size_A + size_B;

	int tailleA = size_A*sizeof(int);
	int tailleB = size_B*sizeof(int);
	int tailleS = size_S*sizeof(int);

	float TimerV;				// GPU timer instructions
	hipEvent_t start, stop;		// GPU timer instructions
	testCUDA(hipEventCreate(&start));		// GPU timer instructions
	testCUDA(hipEventCreate(&stop));		


	
	
	
	testCUDA(hipMalloc(&A_GPU,tailleA));
        testCUDA(hipMalloc(&B_GPU,tailleB));
        testCUDA(hipMalloc(&S_GPU,tailleS));
 
        testCUDA(hipMemcpy(A_GPU,A, tailleA,hipMemcpyHostToDevice));
        testCUDA(hipMemcpy(B_GPU,B, tailleB,hipMemcpyHostToDevice));

	/*On veut que chaque thread ait NEPT éléments de C*/
 	int NB = (size_S + (NTPB*NEPT) -1)/(NTPB*NEPT);	
	int numThreads = NB*NTPB;
        testCUDA(hipEventRecord(start,0));

	kernel<<<NB,NTPB>>>(A_GPU, B_GPU, S_GPU, size_A, size_B, numThreads);
	printf ("NB = %d, NTPB = %d, numthreads = %d\n", NB, NTPB, numThreads);
	
	testCUDA(hipEventRecord(stop,0));
	testCUDA(hipEventSynchronize(stop));
	testCUDA(hipEventElapsedTime(&TimerV,start, stop));

	testCUDA(hipMemcpy(S,S_GPU, tailleS,hipMemcpyDeviceToHost));
	
	hipDeviceSynchronize();
	

	printf("\nExecution time: %f ms\n", TimerV);
        testCUDA(hipFree(A_GPU));
	testCUDA(hipFree(B_GPU));
        testCUDA(hipFree(S_GPU));
}

int main(int argc, char const *argv[]){
	

	FILE * f;
	int *A , *B, *S;
	int A_size, B_size, S_size;
	int i;
	/*================= Recupétation des deux tableaux dans un fichier =================*/

	if (argc < 2) {
        	fprintf( stderr,"Usage: <%s> <JeuxDeDonnees/fichier>\n", argv[0]);
        	return 1;
    	}

	if( (f=fopen(argv[1], "r"))==NULL) {
        	fprintf(stderr,"erreur a la lecture du fichier %s\n", argv[1]);
        	exit(1);
    	}
	
	char ch_a[10] = {0};
    	char ch_b[10] = {0};

	fscanf(f, "%s %s", ch_a,ch_b);
    	A_size = atoi(ch_a);
    	B_size = atoi(ch_b);
	S_size =  A_size + B_size;
	printf("\nSize of A: %d\n",A_size);
	printf("Size of B: %d\n\n",B_size);

	if (SIZE != A_size | NTPB != A_size/NEPT)
	{
		printf ("Make sure : SIZE = %d, NTPB = %d\n", A_size, A_size/NEPT);
		printf ("Please retry !\n");
		return 0;
	}
	
	

	A = (int*) malloc((A_size)*sizeof(int));
	B = (int*) malloc((B_size)*sizeof(int));
	S = (int*) malloc((S_size)*sizeof(int));

	int max = MAX(A_size,B_size);
	int min = MIN(A_size,B_size);

	for(i = 0; i < max; i++){
		if(i < min){
			fscanf(f,"%ld %ld",&A[i],&B[i]);
		}
		else{
			if(min == A_size){
				fscanf(f,"%ld",&B[i]);
			}
			else{
				fscanf(f,"%ld",&A[i]);
			}
		}	
	}
	/*===============================================================================*/


	printGPUCaracteristics();
    	if(A_size == max){
		wrapper(A, B, S, A_size, B_size);
	}
	else{
		wrapper(B, A, S, B_size, A_size);
	}
	
	printf("Given array are \n");
    
	printf("A: ");
	printArray(A, A_size);
	printf("B: ");
	printArray(B, B_size);
    	printf("\nSorted array is \n");
    	printArray(S, S_size);


    
	free(A);
	free(B);
	free(S);


	return 0;
 }
